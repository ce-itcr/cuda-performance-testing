#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <ctime>
#include <iostream>

const int N = 100; // Size of the square matrix

// Kernel to apply Gaussian elimination in parallel
__global__ void gaussianEliminationKernel(float* mat, float* vec, int k) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i > k && i < N) {
        float factor = mat[i * N + k] / mat[k * N + k];
        for (int j = k; j < N; ++j) {
            mat[i * N + j] -= factor * mat[k * N + j];
        }
        vec[i] -= factor * vec[k];
    }
}

// Function to apply Gaussian elimination on GPU
void gaussianEliminationCUDA(float* d_mat, float* d_vec) {
    for (int k = 0; k < N - 1; ++k) {
        int threadsPerBlock = 256;
        int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
        gaussianEliminationKernel <<< blocksPerGrid, threadsPerBlock >>> (d_mat, d_vec, k);
        hipDeviceSynchronize();
    }
}

// Function to solve the upper triangular system on CPU
void solveBackSubstitution(float* mat, float* vec, float* sol) {
    for (int i = N - 1; i >= 0; --i) {
        sol[i] = vec[i];
        for (int j = i + 1; j < N; ++j) {
            sol[i] -= mat[i * N + j] * sol[j];
        }
        sol[i] /= mat[i * N + i];
    }
}

int main() {
    float A[N][N]; // Coefficient matrix on host
    float b[N];    // Independent terms vector on host
    float x[N];    // Solution vector on host

    // Initialization of the matrix and vector
    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            A[i][j] = (i == j) ? 1.0 : 0.0; // Identity matrix
        }
        b[i] = i + 1; // Independent terms vector
    }

    // Memory allocation on the device
    float* d_mat, * d_vec;
    hipMalloc((void**)&d_mat, N * N * sizeof(float));
    hipMalloc((void**)&d_vec, N * sizeof(float));

    // Copy data from host to device
    hipMemcpy(d_mat, A, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_vec, b, N * sizeof(float), hipMemcpyHostToDevice);

    // Measure execution time
    std::clock_t start = std::clock();

    // Apply Gaussian elimination on GPU
    gaussianEliminationCUDA(d_mat, d_vec);

    // Copy results back to host
    hipMemcpy(A, d_mat, N * N * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(b, d_vec, N * sizeof(float), hipMemcpyDeviceToHost);

    // Solve upper triangular system on CPU
    solveBackSubstitution((float*)A, b, x);

    std::clock_t end = std::clock();
    double time = (end - start) / (double)CLOCKS_PER_SEC;

    std::cout << "Execution time (CUDA): " << time << " seconds" << std::endl;

    // Free memory on the device
    hipFree(d_mat);
    hipFree(d_vec);

    return 0;
}
